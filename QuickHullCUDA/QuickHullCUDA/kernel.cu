#include "hip/hip_runtime.h"
#include<time.h>
#include<stdlib.h>
#include<stdio.h>
#include<math.h>
#include<GL/glut.h>
#include <hip/hip_runtime.h>
//#include "timer.h"
#define pi 22/7;

typedef struct {
	double x;
	double y;
} points;

typedef struct {
	double x;
	double y;
	double largest;
} kernel_shared_type;

//points pts[100];
//test case 1
points pts[] = { 3.0, 3.0, 4.0, 6.0, 1.0, 2.0, 5.0, 4.0, 7.0, 3.0, 6.0, 6.0,
		1.0, 1.0, 2.0, 1.0, 6.0, 7.0, 8.0, 8.0, 5.0, 5.0,

		9.0, 4.0, 7.0, 5.0, 1.0, 10.0, 2.0, 7.0, 5.0, 2.0, 2.0, 4.0, 8.0, 6.0,
		3.0, 6.0, 4.0, 4.0, 6.0, 8.0, 5.0, 10.0, 1.0, 5.0, 4.0, 3.0, 8.0, 7.0,

		-2.0, 20.0, 9.0, 0.0, -29.0, -17.0, 17.0, 29.0, 2.0, 3.0, 16.0, 7.0,
		19.0, 7.0, 8.0, 9.0, 19.0, 25.0, 23.0, 4.0, 17.0, 27.0, 18.0, 26.0, 6.0,
		4.0, 12.0, 28.0, 5.0, 25.0, 2.0, 6.0, 28.0, 28.0, 20.0, 15.0, 16.0,
		18.0, 8.0, 12.0, 7.0, 15.0, 3.0, 2.0, 9.0, 1.0, 14.0, 5.0, 4.0, 17.0,
		2.0, 18.0, 7.0, 7.0, 14.0, 9.0, -19.0, -23.0, 12.0, -27.0, -15.0, 12,
		-29, -29, -20, 16, 25.0, 20.0, 14.0, 9.0, 1.0, 3.0, 9.0, 22.0, 6.0, 2.0,
		7.0, 14.0, 19.0, 1.0, 17.0, 14.0, 8.0, 15.0, 25.0, 2.0, 18.0, 1.0, 3.0,
		18.0 };

//test case 2
//points pts[]={-2.0,3.0,-15.0,-2.0,20.0,1.50,6.0,1.0,14.0,0.0,4.0,-2.30,-8.0,1.0,14.0,3.0,-24.0,-3.0,18.0,-2.0,-5.0,3.0,-36.0,23.0,7.0,6.0,16.0,21.0,32.0,15.0,42.0,19.0,22.0,33.0,25.0,35.0,39.0,29.0,28.0,49.0,43.0,17.0,28.0,20.0,37.0,27.0,23.0,34.0,32.0,13.0,36.0,16.0,24.0,35.0,39.0,29.0,15.0,26.0,43.0,34.0,12.0,32.0,31.0,21.0,35.0};	//horizontal points

//test case 3
//points pts[]={2.0,27.0,1.0,-29.0,0.25,-15.62,-0.0,16.45,-1.25,6.0,-1.5,16.0,-1.25,18.0,-1.0,18.75,-2.0,15.0,-1.60,18.0,2.0,-29.0,1.0,-23.0,-2.0,-17.0,-2.0,24.0};//vertical lines

//test case 4
//points pts[]={1.0,7.0,23.0,46.0,27.0,16.0,37.0,18.0,9.0,2.0,35.0,18.0,27.0,21.0,7.0,2.0,23.0,32.0,6.0,3.0,25.0,6.0,28.0,17.0,5.0,38.0,48.0,43.0,28.0,27.0,34.0,47.0,39.0,40.0,35.0,8.0,34.0,32.0,3.0,4.0,3.0,5.0,16.0,9.0,35.0,24.0,1.0,24.0,46.0,29.0,43.0,15.0,32.0,5.0,17.0,16.0,26.0,48.0,29.0,30.0,36.0,24.0,4.0,3.0,12.0,12.0,5.0,8.0,27.0,46.0};

points pt1, pt2, pt11, pt22, smallest, largest;

int i, j, k, r = 0, l2 = 0, count = 0, f_r = 0, screen1 = 0, screen2 = 0, n,
		input = 0;
//int n;
double m, c, temp_c, xx, yy; //m1,c1,m2,c2,temp_c,temp_c1,temp_c2,temp_c3,c_m2,h_m1,h_m2;
points subset1[100], subset2[100], result[100]; //,subsubset1[100],subsubset2[100],result[100];;

/*void getRandomPoints(int n,int xmax,int ymax)
 {

 for (i = 0; i < n; i++)
 {
 xx=rand()%ymax;
 yy=rand()%xmax;
 pts[i].x=xx;
 pts[i].y=yy;
 }
 //points.push( [ xMax /4 + r * Math.cos(theta), yMax/2 + 2 * r * Math.sin(theta) ] )
 }
 /* var phase = Math.random() * Math.PI * 2;
 for (var i = 0; i < numPoint/2; i++) {
 var r =  Math.random()*xMax/4;
 var theta = Math.random() * 1.5 * Math.PI + phase;
 points.push( [ xMax /4 * 3 +  r * Math.cos(theta), yMax/2 +  r * Math.sin(theta) ] )
 }*/

/*void show(points pts[],int n)
 {
 int i;
 printf("N:%d \n",n);*/

void draw(points a, points c) {
	glColor3f(0.0, 0.0, 0.0);
	for (i = 0; i < n; i++) {
		glBegin(GL_POINTS);
		glVertex2d(pts[i].x, pts[i].y);
		glEnd();
	}
	glBegin(GL_LINES);
	{
		glVertex2d(a.x, a.y);
		glVertex2d(c.x, c.y);
	}
	glEnd();
}

points minimum(points *pts, int n) {
	points smallest_x = pts[0];    //pt with smallest x
//	points smallest_y=pts[0];    //pt wit smallest y
	points smallest;

	printf("The Points are: \n");
	for (i = 0; i < n; i++)
		printf("(%.2f,%.2f)\n", pts[i]);

	for (i = 0; i < n; i++) {
		if (pts[i].x < smallest_x.x)
			smallest_x = pts[i];
		else if (pts[i].x == smallest_x.x) {
			if (pts[i].y > smallest_x.y)
				smallest_x = pts[i];
		}
		//if(pts[i].y < smallest_y.y)
		//smallest_y=pts[i];
	}
	printf("(%.2f,%.2f) is smallest x \n", smallest_x);
	//printf("(%.2f,%.2f) is smallest y \n",smallest_y);

	//if((smallest_x.x < smallest_y.x)&&(smallest_x.y < smallest_y.y))    //compares the smallest in x & y and finalize the smallest
	smallest = smallest_x;
	//else
	//smallest=smallest_y;
	//show(pts,n);
	printf("(%.2f,%.2f) is smallest \n", smallest);
	result[f_r++] = smallest;
	return smallest;
}

points maximum(points *pts, int n, points smallest) {
	double distance, largest_distance = 0.0;    //finding the largest distance pt
	double dx, dy;
	points largest;
	for (i = 0; i < n; i++) {
		dx = smallest.x - pts[i].x;    //difference in x
		dy = smallest.y - pts[i].y;    //difference in y
		distance = sqrt((dx * dx) + (dy * dy)); //finding the distance using distance formula
		if (largest_distance < distance) {
			largest_distance = distance;
			largest.x = pts[i].x;
			largest.y = pts[i].y;
		}
	}
	printf("%.2f is largest distance\n", largest_distance);
	printf("(%.2f,%.2f) is largest point\n", largest);
	result[f_r++] = largest;
	return largest;
}

__global__ void subhtKernel(double *d_ab, points *pt, points *a, points *b,
		points *c, int *limit, double *largest) {
	extern __shared__ kernel_shared_type sdata[];
	unsigned int tid = threadIdx.x;
	if (tid < *limit) {
		double dx, dy, t_ac, t_bc, s, area;
		double h;
		dx = a->x - c[tid].x;
		dy = a->y - c[tid].y;
		t_ac = sqrt((dx * dx) + (dy * dy));
		dx = b->x - c[tid].x;
		dy = b->y - c[tid].y;
		t_bc = sqrt((dx * dx) + (dy * dy));
		s = (*d_ab + t_ac + t_bc) / 2;
		area = sqrt(s * (s - *d_ab) * (s - t_ac) * (s - t_bc));
		h = (area * 2) / *d_ab;
		sdata[tid].largest = h;
		sdata[tid].x = c[tid].x;
		sdata[tid].y = c[tid].y;
		__syncthreads();
		for (unsigned int s = *limit / 2; s >= 1; s = s / 2) {
			if (tid < s) {
				if (sdata[tid].largest < sdata[tid + s].largest) {
					sdata[tid].largest = sdata[tid + s].largest;
					sdata[tid].x = sdata[tid + s].x;
					sdata[tid].y = sdata[tid + s].y;
				}
			}
			__syncthreads();
		}
		__syncthreads();
		if (tid == 0) {
			*largest = sdata[0].largest;
			pt->x = sdata[0].x;
			pt->y = sdata[0].y;
		}
	}
}

points subht(points *c, int j, points a, points b) {
	double dx, dy, d_ab, t_ac, t_bc, area, s, largest_h;
	points pt;

	dx = a.x - b.x;    //difference in x
	dy = a.y - b.y;    //difference in y
	d_ab = sqrt((dx * dx) + (dy * dy)); //finding the distance using distance formula

	double *d_d_ab, *d_largest_h;
	int *d_j;
	points *d_pt, *d_a, *d_b, *d_c;

	hipMalloc((void**) &d_d_ab, sizeof(double));
	hipMalloc((void**) &d_largest_h, sizeof(double));
	hipMalloc((void**) &d_j, sizeof(int));
	hipMalloc((void**) &d_pt, sizeof(points));
	hipMalloc((void**) &d_a, sizeof(points));
	hipMalloc((void**) &d_b, sizeof(points));
	hipMalloc((void**) &d_c, sizeof(points) * 100);

	hipMemcpy(d_d_ab, &d_ab, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_j, &j, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_a, &a, sizeof(points), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(points), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, sizeof(points) * 100, hipMemcpyHostToDevice);

	unsigned int shared_size = sizeof(points) * j * j; 

	subhtKernel<<<1, j, shared_size>>>(d_d_ab, d_pt, d_a, d_b, d_c, d_j,
			d_largest_h);

	hipDeviceSynchronize();
	hipMemcpy(&largest_h, d_largest_h, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&pt, d_pt, sizeof(points), hipMemcpyDeviceToHost);
	hipFree(d_d_ab);
	hipFree(d_largest_h);
	hipFree(d_j);
	hipFree(d_pt);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	printf("The maximum height is %.2f and point is (%.2f,%.2f)\n", largest_h,
			pt);
	result[f_r++] = pt;
	return pt;
}

void division(points *pts, int n, points smallest, points largest) {
	m = (double) (largest.y - smallest.y) / (largest.x - smallest.x);    //slope of the line
	c = (double) smallest.y - (m * smallest.x);    //constant of the line
	printf("Slope of the line %.2f \nConstant of the line %.2f \n", m, c);
	printf("(%.2f,%.2f)   (%.2f,%.2f)\n", smallest, largest);

	j = 0;
	k = 0;
#pragma acc parallel loop
	for (i = 0; i < n; i++) {
		temp_c = pts[i].y - (m * pts[i].x);	//y=mx+c
		if (c - temp_c < 0)                    //points above the line
				{
			subset1[j].x = pts[i].x;
			subset1[j].y = pts[i].y;
			j++;
		} else if (c - temp_c > 0)               //points below the line
				{
			subset2[k].x = pts[i].x;
			subset2[k].y = pts[i].y;
			k++;
		}
	}
	printf("Subset1:\n");
	for (i = 0; i < j; i++)
		printf("(%.2f,%.2f)\n", subset1[i]);
	printf("j old:%d\n\n", j);
	printf("k old:%d\n\n", k);
	printf("Subset2:\n");
	for (i = 0; i < k; i++)
		printf("(%.2f,%.2f)\n", subset2[i]);

	pt1 = subht(subset1, j, smallest, largest);
	pt2 = subht(subset2, k, smallest, largest);
}

void divsub1(points *pts, int j, points a, points b, points c) {
	int j2, k2, flag = 0;
	double m1, c1, m2, c2, h_m1, h_m2, c_m2, temp_c1, temp_c2, temp_c3;
	points subsubset1[100], subsubset2[100];

	m1 = (double) (c.y - a.y) / (c.x - a.x);    //slope of the line
	c1 = (double) a.y - (m1 * a.x);    //constant of the line
	printf("Slope of the line %.2f \nConstant of the line %.2f \n", m1, c1);
	printf("(%.2f,%.2f) , (%.2f,%.2f)\n", a, c);

	m2 = (double) (c.y - b.y) / (c.x - b.x);    //slope of the line
	c2 = (double) b.y - (m2 * b.x);    //constant of the line
	printf("Slope of the line %.2f \nConstant of the line %.2f \n", m2, c2);
	printf("(%.2f,%.2f) , (%.2f,%.2f)\n", b, c);

	h_m1 = (b.y - a.y) / (b.x - a.x);
	h_m2 = -1 / (h_m1);
	c_m2 = (double) c.y - (h_m2 * c.x);
	printf("Slope of the line %.2f \nConstant of the line %.2f \n", h_m2, c_m2);

	j2 = 0;
	k2 = 0;
	printf("J:%d\n\n", j);
#pragma acc parallel loop
	for (i = 0; i < j; i++) {
		temp_c3 = pts[i].y - (h_m2 * pts[i].x);	//line hc
		temp_c2 = pts[i].y - (m2 * pts[i].x);	//line bc
		temp_c1 = pts[i].y - (m1 * pts[i].x);	//y=mx+c line a

		printf("point is:(%2.f,%2.f)\n", pts[i]);
		if ((pts[i].x - c.x) != 0)		//infinte line
				{
			if (h_m2 > 0)		//positive slope
					{
				if (c_m2 - temp_c3 < 0)	//above the + line
						{
					printf("1-111111111\n");
					if (c1 - temp_c1 < 0)	//above the line
							{
						printf("1-222222\n");
						subsubset1[j2].x = pts[i].x;
						subsubset1[j2].y = pts[i].y;
						j2++;
						//flag=1;
					}
				} else if (c_m2 - temp_c3 > 0)		//below the line +line
						{
					printf("1-3333333\n");
					{
						if (m2 < 0)	//negative slope
								{
							if (c2 - temp_c2 < 0)	//above the line
									{
								printf("1-44444111111\n");
								subsubset2[k2].x = pts[i].x;
								subsubset2[k2].y = pts[i].y;
								k2++;
								//flag=1;
							}
						} else if (m2 > 0)	//positive slope
								{
							if (c2 - temp_c2 > 0)	//below the line
									{
								printf("1-4444422222\n");
								subsubset2[k2].x = pts[i].x;
								subsubset2[k2].y = pts[i].y;
								k2++;
								//flag=1;
							}
						}
					}
					/*else

					 {
					 */
				}
			}

			else if (h_m2 < 0)		//negative slope
					{
				if (c_m2 - temp_c3 > 0)	//below the line -ve line
						{
					printf("1-55555555\n");
					if (m1 > 0)	//positive slope
							{
						if (c1 - temp_c1 < 0)	//above the line
								{
							printf("1-66666666\n");
							subsubset1[j2].x = pts[i].x;
							subsubset1[j2].y = pts[i].y;
							j2++;
						}
					} else if (m1 < 0) {
						if (c1 - temp_c1 > 0)	//below the line
								{
							printf("1-66666666\n");
							subsubset1[j2].x = pts[i].x;
							subsubset1[j2].y = pts[i].y;
							j2++;
						}
					}
				} else if (c_m2 - temp_c3 < 0)	//above the line -ve line
						{
					printf("1-777777777\n");
					if (c2 - temp_c2 < 0)	//above the line
							{
						printf("1-88888888\n");
						subsubset2[k2].x = pts[i].x;
						subsubset2[k2].y = pts[i].y;
						k2++;
					}
				}
			} else if (h_m2 == 0)	//horizontal line
					{
				if (c.y - pts[i].y < 0)	//above the horizontal line
						{
					if (c1 - temp_c1 < 0)	//above the line
							{
						printf("1-999999\n");
						subsubset1[j2].x = pts[i].x;
						subsubset1[j2].y = pts[i].y;
						j2++;
					}
				} else if (c.y - pts[i].y < 0)		//below the horizontal line
						{
					if (c2 - temp_c2 > 0)	//below the line
							{
						printf("1-1010101\n");
						subsubset2[k2].x = pts[i].x;
						subsubset2[k2].y = pts[i].y;
						k2++;
					}
				}
			}
		} else {
			if (c.x - pts[i].x < 0) //above the vertical line
					{
				if (c1 - temp_c1 < 0)		//above
						{
					printf("1-12121212\n");
					subsubset1[j2].x = pts[i].x;
					subsubset1[j2].y = pts[i].y;
					j2++;
				}
			} else if (c.x - pts[i].x > 0)		//below the vertical line
					{
				if (c2 - temp_c2 > 0)		//below
						{
					printf("1-13131313\n");
					subsubset2[k2].x = pts[i].x;
					subsubset2[k2].y = pts[i].y;
					k2++;
				}
			}
		}
	}

	if (j2 != 0) {
		printf(" subset1:\n");
		for (i = 0; i < j2; i++)
			printf("(%.2f,%.2f)\n", subsubset1[i]);
		pt11 = subht(subsubset1, j2, a, c);
		divsub1(subsubset1, j2, a, c, pt11);
	} else {
		draw(a, c);
	}

	if (k2 != 0) {
		printf(" subset2:\n");
		for (i = 0; i < k2; i++)
			printf("(%.2f,%.2f)\n", subsubset2[i]);
		pt22 = subht(subsubset2, k2, c, b);
		divsub1(subsubset2, k2, c, b, pt22);
	} else {
		draw(c, b);
	}

}

void divsub2(points *pts, int k, points a, points b, points c) {
	int j2, k2;
	double m1, c1, m2, c2, h_m1, h_m2, c_m2, temp_c1, temp_c2, temp_c3;
	points subsubset1[100], subsubset2[100];

	m1 = (double) (c.y - a.y) / (c.x - a.x);    //slope of the line
	c1 = (double) a.y - (m1 * a.x);    //constant of the line
	printf("Slope of the line %.2f \nConstant of the line %.2f \n", m1, c1);
	printf("(%.2f,%.2f) , (%.2f,%.2f)\n", a, c);

	m2 = (double) (c.y - b.y) / (c.x - b.x);    //slope of the line
	c2 = (double) b.y - (m2 * b.x);    //constant of the line
	printf("Slope of the line %.2f \nConstant of the line %f \n", m2, c2);
	printf("(%.2f,%.2f) , (%.2f,%.2f)\n", b, c);

	h_m1 = (b.y - a.y) / (b.x - a.x);
	h_m2 = -1 / (h_m1);
	c_m2 = (double) c.y - (h_m2 * c.x);
	printf("Slope of the line %.2f \nConstant of the line %.2f \n", h_m2, c_m2);

	j2 = 0;
	k2 = 0;
	for (i = 0; i < k; i++) {
		temp_c3 = pts[i].y - (h_m2 * pts[i].x);	//line hc
		temp_c2 = pts[i].y - (m2 * pts[i].x);	//line bc
		temp_c1 = pts[i].y - (m1 * pts[i].x);	//y=mx+c line ac
		printf("temp_c2,(pt),index==%f,(%.2f,%.2f),%d\n", temp_c2, pts[i], i);

		if ((pts[i].x - c.x) != 0)		//infinte line
				{
			if (h_m2 > 0)		//positive slope
					{
				if (c_m2 - temp_c3 < 0)		//above the +line
						{
					printf("2-11111 \n");
					if (c1 - temp_c1 > 0)	//below the line
							{
						printf("2-222222 \n");
						subsubset1[j2].x = pts[i].x;
						subsubset1[j2].y = pts[i].y;
						j2++;
					}
				} else if (c_m2 - temp_c3 > 0)			//below the +line
						{
					printf("2-333333 \n");
					if (c2 - temp_c2 > 0)	//below the line
							{
						printf("2-44444444444\n");
						subsubset2[k2].x = pts[i].x;
						subsubset2[k2].y = pts[i].y;
						k2++;
					}
				}
			} else if (h_m2 < 0)		//negative slope
					{
				printf("2-55555555 \n");
				if (c_m2 - temp_c3 > 0)		//below the -line
						{
					printf("2-66666 \n");
					if (c1 - temp_c1 > 0)	//below the line
							{
						subsubset1[j2].x = pts[i].x;
						subsubset1[j2].y = pts[i].y;
						j2++;
					}
				} else if (c_m2 - temp_c3 < 0)		//above the -line
						{
					printf("2-77777777 \n");
					if (m2 < 0)		//negative slope
							{
						if (c2 - temp_c2 < 0)	//above the line
								{
							printf("2-88881111\n");
							subsubset2[k2].x = pts[i].x;
							subsubset2[k2].y = pts[i].y;
							k2++;
						}
					} else if (m2 > 0)		//positive slope
							{
						if (c2 - temp_c2 > 0)	//below the line
								{
							printf("2-888822222\n");
							subsubset2[k2].x = pts[i].x;
							subsubset2[k2].y = pts[i].y;
							k2++;
						}
					}
				}
			} else if (h_m2 == 0)	//horizontal line
					{
				if (c.y - pts[i].y > 0)	//below the horizontal line
						{
					if (c1 - temp_c1 > 0)	//below the line
							{
						printf("2-999999\n");
						subsubset1[j2].x = pts[i].x;
						subsubset1[j2].y = pts[i].y;
						j2++;
					}
				} else if (c.y - pts[i].y < 0)		//above the horizontal line
						{
					if (c2 - temp_c2 > 0)	//below the line
							{
						printf("2-1010101\n");
						subsubset2[k2].x = pts[i].x;
						subsubset2[k2].y = pts[i].y;
						k2++;
					}
				}
			}
		} else {
			if (c.x - pts[i].x > 0) //below the vertical line
					{
				if (c1 - temp_c1 > 0)		//below the line
						{
					printf("2-12121212\n");
					subsubset1[j2].x = pts[i].x;
					subsubset1[j2].y = pts[i].y;
					j2++;
				}
			} else if (c.x - pts[i].x < 0)		//above the vertical line
					{
				if (c2 - temp_c2 < 0)		//above the line
						{
					printf("1-13131313\n");
					subsubset2[k2].x = pts[i].x;
					subsubset2[k2].y = pts[i].y;
					k2++;
				}
			}
		}
	}

	printf("j2:%d\n", j2);
	printf("k2:%d\n", k2);

	if (j2 != 0) {
		printf("Subsubset1:\n");
		for (i = 0; i < j2; i++)
			printf("(%.2f,%.2f)\n", subsubset1[i]);
		pt11 = subht(subsubset1, j2, a, c);
		divsub2(subsubset1, j2, a, c, pt11);
	} else {
		draw(a, c);
	}

	if (k2 != 0) {
		printf("Subsubset2:\n");
		for (i = 0; i < k2; i++)
			printf("(%.2f,%.2f)\n", subsubset2[i]);
		pt22 = subht(subsubset2, k2, c, b);
		divsub2(subsubset2, k2, c, b, pt22);
	} else {
		draw(c, b);
	}
}

int main1() {
	/*clock_t start = clock(), stop, t;*/
	double elapsed, xx = 0, yy = 0;

	//clock_t t;
	/*int i,ti;
	 t=clock();
	 ti=(unsigned int)t;
	 for(i=0;i<50;i++)
	 {
	 ti++;
	 srand(ti);
	 xx= rand()%50+1;
	 yy= rand()%50+1;
	 pts[i].x=xx;
	 pts[i].y=yy;
	 }
	 */

	float elapsedTime;
	
hipEvent_t start,stop;
	
hipEventCreate(&start);
	
hipEventCreate(&stop);
	
hipEventRecord(start,0);
	n = (sizeof(pts) / sizeof(double)) / 2;

	smallest = minimum(pts, n);
	largest = maximum(pts, n, smallest);
	division(pts, n, smallest, largest);

	divsub1(subset1, j, smallest, largest, pt1);
	divsub2(subset2, k, smallest, largest, pt2);

	printf("final hull");
	for (i = 0; i < f_r; i++)
		printf("(%.2f,%.2f)\n", result[i]);

	hipDeviceSynchronize();
	
hipEventRecord(stop,0);
	
hipEventSynchronize(stop);
	
hipEventElapsedTime(&elapsedTime,start,stop);
	
printf("Time Taken = %f ms",elapsedTime);

	//stop = clock();
	//elapsed = (double) (stop - start) * 1000.0 / CLOCKS_PER_SEC;
	//printf("Time elapsed in ms: %.2f", elapsed);

	return 0;
}

void keys(unsigned char key, int x, int y) {
	/*if(key=='s')
	 {
	 screen1=1;
	 glutPostRedisplay();
	 }*/
	if (key == 'a') {
		screen2 = 1;
		glutPostRedisplay();
	}
	/*if(key=='i')
	 {
	 input=1;
	 glutPostRedisplay();
	 }*/

}

void mydisplay() {
	glClear(GL_COLOR_BUFFER_BIT);
	/*if(screen1==1)
	 {
	 show(pts,n);
	 screen1=0;
	 }*/
	if (screen2 == 1) {
		//show(pts,n);
		main1();
		screen2 = 0;
	}
	/*if(input==1)
	 {
	 getRandomPoints(20,25.00,25.00);
	 input=0;
	 }*/
//screen=0;
	glFlush();
}

void init() {
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glPointSize(4.0);
	gluOrtho2D(-60, 60, -60, 60);
	glClearColor(1.0, 1.0, 1.0, 1.0);
	glColor3f(0.0, 0.0, 0.0);
	glMatrixMode(GL_MODELVIEW);
	//  glLoadIdentity();
}

int main(int argc, char **argv) {
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);
	glutInitWindowSize(600, 600);
	glutInitWindowPosition(10, 10);
	glutCreateWindow("simple");
	glutKeyboardFunc(keys);
	glutDisplayFunc(mydisplay);
	init();
	glutMainLoop();
}

