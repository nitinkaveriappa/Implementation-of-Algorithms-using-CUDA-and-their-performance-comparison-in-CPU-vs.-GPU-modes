#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
//#include<windows.h>
//#include<string.h>
#include<gl/glut.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
//#include<math.h>
//using namespace std;
int map[500][500];
int screen=0,number1;

void myinit(void)
{
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(-50,50,0,200,-10,10);
	glClearColor(0.0,0.0,0.0,1.0);
	 glColor3f(1.0, 0.0, 0.0);
	//glColor3f(1.0,1.0,1.0);
	glMatrixMode(GL_MODELVIEW);
	
}

__global__ void diamondKernel(int *d_map, int *d_minimum, int *d_s_size,hiprandState *devState) {
	int left, right, up, down, val[4], avg;

	int row = (blockIdx.y * blockDim.y + threadIdx.y) * (*d_s_size)	+ (*d_minimum);
	//row += blockDim.y * gridDim.y;
	int col = (blockIdx.x * blockDim.x + threadIdx.x) * (*d_s_size)	+ (*d_minimum);
	//col += blockDim.x * gridDim.x;

	if(row < (*d_s_size - *d_minimum))
	{
		if(col < (*d_s_size - *d_minimum)) {
//		hiprandState s;
		int seed = row * col;
		hiprand_init(seed, 0, 0, (devState + row)+col);
		left = row - *d_minimum;
		right = row + *d_minimum;
		up = col - *d_minimum;
		down = col + *d_minimum;

		// the four corner values
		val[0] = *((d_map + left) + up);   // upper left
		val[1] = *((d_map + left) + down); // lower left
		val[2] = *((d_map + right) + up);  // upper right
		val[3] = *((d_map + right) + down);  // lower right

		avg = (val[0] + val[1] + val[2] + val[3]) / 4;
		//srand(ti);
		int temp = (int)((hiprand_uniform((devState + row)+col))*10000)%25;
		//printf("%d\n",number1);
		//atomicAdd((d_map + row) + col,(avg+temp));
		*((d_map + row) + col) = avg + temp;
	}
	}
}

void diamond(int size, int minimum, int maximum, int iterations) {
//	clock_t t;
	int i,j;
	int s_size;
//	t = clock();
	//ti = (unsigned int) t;
//	int seed = (unsigned int) t;
	hiprandState *devState;
	//printf("diamond\n");
	s_size = (1 << iterations) + 1;

	int *d_map, *d_minimum, *d_s_size;
	hipMalloc((void**)&d_map, sizeof(int) * 500 * 500);
	hipMalloc((void**)&d_minimum, sizeof(int));
	hipMalloc((void**)&d_s_size, sizeof(int));
	hipMalloc((void**)&devState, (s_size - minimum) * (s_size - minimum) * sizeof(hiprandState));
	hipMemcpy(d_map, map, sizeof(map), hipMemcpyHostToDevice);
	hipMemcpy(d_minimum, &minimum, sizeof(minimum), hipMemcpyHostToDevice);
	hipMemcpy(d_s_size, &s_size, sizeof(s_size), hipMemcpyHostToDevice);

	dim3 dimGrid(1, 1);
	dim3 blockDimensional(s_size - minimum, s_size - minimum);

	diamondKernel<<<1, blockDimensional>>>(d_map, d_minimum, d_s_size,devState);
	hipDeviceSynchronize();

	hipMemcpy(map, d_map, sizeof(int) * 500 * 500, hipMemcpyDeviceToHost);
	//	printf(hipGetErrorString(hipGetLastError()));

	hipFree(d_map);
	hipFree(d_minimum);
	hipFree(d_s_size);
	hipFree(devState);

//pc
//	for (x = minumum; x < (s_size - minumum); x += size) {
//		for (y = minumum; y < (s_size - minumum); y += size) {
//			left = x - minumum;
//			right = x + minumum;
//			up = y - minumum;
//			down = y + minumum;
//
//			// the four corner values
//			val1 = map[left][up];   // upper left
//			val2 = map[left][down]; // lower left
//			val3 = map[right][up];  // upper right
//			val4 = map[right][down];  // lower right
//
//			avg = (val1 + val2 + val3 + val4) / 4;
//			srand(ti);
//			number1 = rand() % 25;
//			//printf("%d\n",number1);
//			map[x][y] = avg + number1;
//			ti++;
//
//		}
//	}

	for( i=0;i<=maximum;i++)
	 {
	 for( j=0;j<=maximum;j++)
	 {
	 printf("   %d",map[i][j]);
	 }
	 printf("\n");
	 }
	/*printf(
			"\n***************************************************************************");*/
	/*for (int i = 0; i <= maximum; i++) {
		for (int j = 0; j <= maximum; j++) {
			printf("   %d", map[i][j]);
		}
		printf("\n");
	}
	printf(
			"\n***************************************************************************");*/

}


void squareStepEven(int min,int size,int max,int iterations)
 {
	 clock_t t;
	int ti,x,y,s_size,left,right,up,down,val1,val2,val3,val4,avg,i,j;
	t=clock();
	ti=(unsigned int)t;
	// printf("square1\n");
	  s_size=(1<<iterations)+1 ;
          for ( x = min; x < s_size; x += size)
		  {
               for (y = 0; y < s_size; y += size)
			   {
                    if (y == max)
					{
                         map[x][y] = map[x][0];
                         continue;
                    }

                     left = x - min;
                     right = x + min;
                     down = y + min;
                     up = 0;

                    if (y == 0)
					{
                         up = max-min;
                    } else 
					{
                         up = y - min;
                    }

                    // the four corner values
                     val1 = map[left][y]; // left
                     val2 = map[x][up];   // up
                     val3 = map[right][y];// right   
				     val4 = map[x][down]; // down
			    avg=(val1+val2+val3+val4)/4;
			   srand (ti);
				number1 = rand() % 25;
				//printf("%d\n",number1);
                    map[x][y]=avg+number1;
					ti++;
			   }

		  }
		  
		/*  for(i=0;i<=max;i++)
	{
		for(j=0;j<=max;j++)
			{
				printf("   %d",map[i][j]);
			}
		printf("\n");
		  }*/
 }
 void squareStepOdd(int min, int size,int max,int iterations)
 {
	 clock_t t;
	int ti,x,y,i,j,s_size,left,right,down,up,avg,val1,val2,val3,val4;
	t=clock();
	ti=(unsigned int)t;
	  s_size=(1<<iterations)+1 ;
	// printf("square2\n");
	 printf("size=%d \n",size);
          for ( x = 0; x < s_size; x += size)
		  {
               for ( y = min; y < s_size; y += size)
			   {
                    if (x == max)
					{
                         map[x][y] = map[0][y];
                         continue;
					}

                     left = 0;
                     right = x + min;
                     down = y + min;
                     up = y - min;

                    if (x == 0)
					{
                         left = max - min;
                    } else
					{
                         left = x - min;
                    }

                    // the four corner values
                     val1 = map[left][y]; // left
                     val2 = map[x][up];   // up
                     val3 = map[right][y];// right
                     val4 = map[x][down]; // down
					 avg=(val1+val2+val3+val4)/4;
					srand (ti);
					number1 = rand() % 25;
					//printf("%d\n",number1);
                    map[x][y]=avg+number1;
					ti++;
			   }
		  }
		 
	/*	  for( i=0;i<=max;i++)
	{
		for( j=0;j<=max;j++)
			{
				printf("   %d",map[i][j]);
			}
		printf("\n");
		  }*/
 }
 void makeMap(int iterations,int seed)
{
	clock_t start=clock(),stop;
	double elapsed;
	//printf("this s construction\n");
	int size,minCoordinate,maxIndex,i,j;
	//int **map;
	size=(1<<iterations)+1;
	//int **map=new int *[size];
	printf("size is %d \n",size);
	//for( i=0;i<size;i++)
		//map[i]=new int[size];
	for( i=0;i<size;i++)
		for( j=0;j<size;j++)
			map[i][j]=0;
	 maxIndex=size-1;
	 printf("maxIndex:%d",maxIndex);

	 map[0][0]=seed;
	 map[0][maxIndex]=seed;
	 map[maxIndex][maxIndex]=seed;
	 map[maxIndex][0]=seed;
	/* for( i=0;i<size;i++)
	{
		for( j=0;j<size;j++)
			{
				printf("   %d",map[i][j]);
			}
		printf("\n");
	 }*/
	
	for( i=1;i<=iterations;i++)
	{
		minCoordinate=maxIndex>>i;
		size=minCoordinate<<1;
		//printf("size=%d",size);
		diamond(size,minCoordinate,maxIndex,iterations);
		//printf("size=%d",size);
		squareStepEven(minCoordinate,size,maxIndex,iterations);
		//printf("size=%d",size);
		squareStepOdd(minCoordinate,size,maxIndex,iterations);
	}
	printf("end\n");
	/*for( i=0;i<=maxIndex;i++)
	{
		for( j=0;j<=maxIndex;j++)
			{
				printf("   %d",map[i][j]);
			}
		printf("\n");
	}*/
	stop=clock();
	elapsed = (double)(stop - start) * 1000.0 / CLOCKS_PER_SEC;
    printf("Time elapsed in ms: %.2f", elapsed);
 }




 void keys(unsigned char key,int x,int y)
 {
	// int **map;
	 if(key=='a')
	 {
		 screen=1;
		 glutPostRedisplay();
	 }
 }

 void display()
 {
	//int ** map;
	int size,s,i,j;
	GLfloat x,z,step;
	

	
	//int s=50;
	 if(screen==1)
	 {
	 //map=makeMap(5,50);
  //glColor3fv(c);
    glClear(GL_COLOR_BUFFER_BIT|GL_DEPTH_BUFFER_BIT);
	size=(1<<5)+1;
	makeMap(5,50);
	 s=size-1;
	 step = (GLfloat)2*s / size;
     x = -(GLfloat)s;
     z = -(GLfloat)s;
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
    glBegin(GL_QUADS);
    for ( i = 0; i < s; i++) 
	{
        for ( j = 0; j < s; j++) 
		{
		glColor3f(i/50.0,j/16000.0,map[i][j]/16000.00);
            glVertex3f(x, map[i][j], z);
            glVertex3f(x, map[i][j+1], z+step);
            glVertex3f(x+step, map[i+1][j+1], z+step);
            glVertex3f(x+step, map[i+1][j], z);

            z += step;
        }
        x += step;
        z = -(GLfloat)s;
    }
	 }
    glEnd();
	screen=0;
	glutPostRedisplay();
	glFlush();
 }

 int main(int argc,char ** argv)
{
    glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_SINGLE|GLUT_RGB|GLUT_DEPTH);
	glutInitWindowSize(500,500);
	glutCreateWindow("terrains");
	glutKeyboardFunc(keys);
	glutDisplayFunc(display);
	glEnable(GL_DEPTH_TEST);
	myinit();
	glutMainLoop();
}