#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
//#include "timer.h"


#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} \
}

#define WORK_SIZE 100 * sizeof(char)

#define match 2
#define mismatch -1
/*char strA[100] = { 'a', 'g', 'c', 'a', 'c', 'a', 'c', 'a', 'a', 'g', 'c', 'a',
		'c', 'a', 'c', 'a', 'a', 'g', 'c', 'a', 'c', 'a', 'c', 'a', 'a', 'g',
		'c', 'a', 'c', 'a', 'c', 'a', 'a', 'g', 'c', 'a', 'c', 'a', 'c', 'a',
		'\0' };
char strB[100] = { 'a', 'g', 'c', 'a', 'c', 'a', 'a', 'a', 'a', 'c', 'a', 'c',
		'a', 'c', 't', 'a', 'a', 'c', 'a', 'c', 'a', 'c', 't', 'a', 'a', 'c',
		'a', 'c', 'a', 'c', 't', 'a', 'a', 'c', 'a', 'c', 'a', 'c', 't', 'a',
		'\0' };
*/
char strA[100] = { 'a', 'c', 'a', 'c', 't', 'a', 'g', 'a', 'c', 't', 'a', '\0' };
char strB[100] = { 'a', 'g', 'c', 'a', 'c', 'a', 'g', 'a', 'c', 'i', 'o', '\0' };


char strAF[100], strBF[100];
int i, j, k, l, m, lenA, lenB, compval;
int maximum = 0, mati, matj;
int SmatArray[100][100], AmatArray[100][100];

typedef struct {
	int x;
	int y;
} points;
points allignPts[100];

__global__ void localAllign(points *p_allignPts, char *p_strAF, char *p_strBF,
		char *p_strA, char *p_strB, int *p_m) {
	int index = threadIdx.x;
	if (((p_allignPts[index].x - p_allignPts[index + 1].x) == 1)
			&& ((p_allignPts[index].y - p_allignPts[index + 1].y) == 1)) {
		p_strAF[*p_m - index] = p_strA[(p_allignPts[index].x) - 1];
		p_strBF[*p_m - index] = p_strB[(p_allignPts[index].y) - 1];
	} else if (((p_allignPts[index].x - p_allignPts[index + 1].x) == 0)
			&& ((p_allignPts[index].y - p_allignPts[index + 1].y) == 1)) {
		p_strAF[*p_m - index] = '-';
		p_strBF[*p_m - index] = p_strB[(p_allignPts[index].y) - 1];
	} else if (((p_allignPts[index].x - p_allignPts[index + 1].x) == 1)
			&& ((p_allignPts[index].y - p_allignPts[index + 1].y) == 0)) {
		p_strAF[*p_m - index] = p_strA[(p_allignPts[index].x) - 1];
		p_strBF[*p_m - index] = '-';
	}
}

void scoringmat(char strA[100], char strB[100], int lenA, int lenB) {
	printf("Scoring Matrix");
	for (i = 0; i < lenA; i++) {
		SmatArray[0][i] = 0;
	}
	for (i = 0; i < lenB; i++) {
		SmatArray[i][0] = 0;
	}

	compval = 0;
	for (i = 1; i <= lenA; i++) {
		for (j = 1; j <= lenB; j++) {
			if (strA[i - 1] == strB[j - 1]) {
				compval = (SmatArray[i - 1][j - 1] + match);
			}
			if (compval < (SmatArray[i - 1][j] + mismatch)) {
				compval = (SmatArray[i - 1][j] + mismatch);
			}
			if (compval < (SmatArray[i][j - 1] + mismatch)) {
				compval = (SmatArray[i][j - 1] + mismatch);
			} else if (strA[i - 1] != strB[j - 1]) {
				compval = (SmatArray[i - 1][j - 1] + mismatch);
			}
			if (compval < (SmatArray[i - 1][j] + mismatch)) {
				compval = (SmatArray[i - 1][j] + mismatch);
			}
			if (compval < (SmatArray[i][j - 1] + mismatch)) {
				compval = (SmatArray[i][j - 1] + mismatch);
			}
			if (compval < 0) {
				compval = 0;
			}
			SmatArray[i][j] = compval;
			compval = 0;
		}
	}
	printf("\n");
	printf("   0");
	for (i = 0; i <= lenB; ++i) {
		printf("  %c", strB[i]);
	}
	printf("\n");
	for (i = 0; i < lenA; ++i) {
		if (i < 1) {
			printf("0");
		}
		if (i > 0) {
			printf("%c", strA[i - 1]);
		}
		for (j = 0; j < lenB; ++j) {
			printf("%3i", SmatArray[i][j]);
		}
		printf("\n");
	}
}

void allignmat() {
	for (i = 0; i < lenA; i++) {
		for (j = 0; j < lenB; j++) {
			if (SmatArray[i][j] > maximum) {
				maximum = SmatArray[i][j];
				mati = i;
				matj = j;
			}
		}
	}
	printf("matrix value=%d and cell value=%d %d\n", maximum, mati, matj);
	printf("\n");
	i = mati;
	j = matj;
	//printf("matrix value=%d\n",SmatArray[i][j]);
	k = 0;
	allignPts[k].x = i;
	allignPts[k].y = j;
	k++;
	while (SmatArray[i][j] != 0) {
		i = mati;
		j = matj;
		if ((SmatArray[i - 1][j - 1] >= SmatArray[i - 1][j])
				&& (SmatArray[i - 1][j - 1] >= SmatArray[i][j - 1])) {
			maximum = SmatArray[i - 1][j - 1];
			//printf("matrix value=%d\n ",max);
			allignPts[k].x = i - 1;
			allignPts[k].y = j - 1;
			k++;
			mati = i - 1;
			matj = j - 1;
		} else if ((SmatArray[i - 1][j] > SmatArray[i - 1][j - 1])
				&& (SmatArray[i - 1][j] > SmatArray[i][j - 1])) {
			maximum = SmatArray[i - 1][j];
			//printf("matrix value=%d\n ",max)l;
			allignPts[k].x = i - 1;
			allignPts[k].y = j;
			k++;
			mati = i - 1;
			matj = j;
		} else if ((SmatArray[i][j - 1] > SmatArray[i - 1][j - 1])
				&& (SmatArray[i][j - 1] > SmatArray[i - 1][j])) {
			maximum = SmatArray[i][j - 1];
			//printf("matrix value=%d\n ",max);
			allignPts[k].x = i;
			allignPts[k].y = j - 1;
			k++;
			mati = i;
			matj = j - 1;
		}
		i = i--;
		j = j--;
	}

	for (i = 0; i <= k; i++) {
		printf("(%d,%d)\n", allignPts[i]);
	}
	l = k - 1;
	printf("\nAllignment Matrix\n");
	printf("   0");
	for (i = 0; i <= lenB; ++i) {
		printf("  %c", strB[i]);
	}
	printf("\n");
	for (i = 0; i < lenA; ++i) {
		if (i < 1) {
			printf("0");
		}
		if (i > 0) {
			printf("%c", strA[i - 1]);
		}

		for (j = 0; j < lenB; ++j) {
			if (allignPts[k].x == i && allignPts[k].y == j) {
				printf("%3i", SmatArray[i][j]);
				k--;
			} else
				printf("  0");
		}
		printf("\n");
	}

}

int main(void) {
	clock_t start1,stop1;
	double elapsed1;
	printf("\tSMITH WATERMAN C PROGRAM\n\n");
	//printf("string1=agcacaca\n");
	//printf("string2=acacacta\n\n");

	lenA = strlen(strA) + 1;
	lenB = strlen(strB) + 1;
	//StartTimer();
	float elapsedTime;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	start1=clock();
	scoringmat(strA, strB, lenA, lenB);
	allignmat();

	m = l;
	points *d_allignPts;
	int *d_m;
	char *d_strA, *d_strB, *d_str_AF, *d_strBF;
	CUDA_CHECK_RETURN(hipMalloc((void**) &d_allignPts, sizeof(allignPts)));
	CUDA_CHECK_RETURN(hipMalloc((void**) &d_strA, WORK_SIZE));
	CUDA_CHECK_RETURN(hipMalloc((void**) &d_strB, WORK_SIZE));
	CUDA_CHECK_RETURN(hipMalloc((void**) &d_str_AF, WORK_SIZE));
	CUDA_CHECK_RETURN(hipMalloc((void**) &d_strBF, WORK_SIZE));
	CUDA_CHECK_RETURN(hipMalloc((void**) &d_m, sizeof(int)));
	CUDA_CHECK_RETURN(hipMemcpy(d_allignPts, allignPts, sizeof(allignPts),
			hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_strA, strA, WORK_SIZE, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_strB, strB, WORK_SIZE, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_str_AF, strAF, WORK_SIZE, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_strBF, strBF, WORK_SIZE, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_m, &m, sizeof(int), hipMemcpyHostToDevice));

	localAllign<<<1, m+1>>>(d_allignPts, d_str_AF, d_strBF, d_strA, d_strB, d_m);

	CUDA_CHECK_RETURN(hipMemcpy(strAF, d_str_AF, WORK_SIZE, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(strBF, d_strBF, WORK_SIZE, hipMemcpyDeviceToHost));
	hipDeviceSynchronize();
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	stop1=clock();
	elapsed1=(double)(stop1-start1)*1000.0/CLOCKS_PER_SEC;
	printf("\nTime elapsed in ms: %.2f\n",elapsed1);
	//printf("Time Taken = %f ms",elapsedTime);
	printf("\nOptimal Local Alignment of Sequences A & B\nSeqA: ");
	for (i = 0; i <= m+1; i++) {
		printf("%c", strAF[i]);
	}
	printf("\nSeqB: ");
	for (i = 0; i <= m+1; i++) {
		printf("%c", strBF[i]);
	}
	printf("\n");

	//double runtime = GetTimer();
	//printf(" total: %f s\n", runtime / 1000);

	CUDA_CHECK_RETURN(hipFree(d_m));
	CUDA_CHECK_RETURN(hipFree(d_allignPts));
	CUDA_CHECK_RETURN(hipFree(d_strA));
	CUDA_CHECK_RETURN(hipFree(d_strB));
	CUDA_CHECK_RETURN(hipFree(d_str_AF));
	CUDA_CHECK_RETURN(hipFree(d_strBF));
	hipDeviceReset();
	return 0;
}